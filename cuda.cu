/*
 *   Copyright (c) 2021 
 *   All rights reserved.
 */
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>

bool InitCUDA() {
    int count = 0;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "This is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; ++i) {
        struct hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x .\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}